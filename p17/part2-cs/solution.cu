#include <climits>
#include <cmath>
#include <hip/hip_runtime.h>

#include <stdio.h>

#define CUDA_CHECK(call)                                                       \
  do {                                                                         \
    hipError_t error = call;                                                  \
    if (error != hipSuccess) {                                                \
      printf("CUDA error at %s:%d: %s\n", __FILE__, __LINE__,                  \
             hipGetErrorString(error));                                       \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

void printDeviceInfo() {
  int deviceCount;
  CUDA_CHECK(hipGetDeviceCount(&deviceCount));

  if (deviceCount == 0) {
    printf("No CUDA devices found!\n");
    exit(1);
  }

  printf("Found %d CUDA devices:\n", deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, i));
    printf("Device %d: %s\n", i, prop.name);
    printf("  Compute capability: %d.%d\n", prop.major, prop.minor);
    printf("  Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("  Max blocks per multiprocessor: %d\n",
           prop.maxBlocksPerMultiProcessor);
    printf("  Number of multiprocessors: %d\n", prop.multiProcessorCount);
  }
  printf("\n");
}

__device__ long long executeProgram(long long A) {
  // Program constants
  long long res = 0LL;
  long long B = 0LL;
  long long C = 0LL;

  do {
    B = A % 8;
    B ^= 7;
    C = A >> B;
    B ^= 7;
    B = B ^ C;
    A >>= 3;
    res += B % 8;
    res *= 10;
  } while (A);

  return res / 10;
}

__global__ void searchKernel(long long start_value, long long target,
                             int values_per_thread, bool *found,
                             long long *result) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  long long my_start = start_value + (long long)idx * values_per_thread;

  for (int i = 0; i < values_per_thread; i++) {
    long long current = my_start + i;
    long long computed = executeProgram(current);

    if (computed == target) {
      *found = true;
      *result = current;
      return;
    }
  }
}

int main() {
  printDeviceInfo();

  const long long target = 2417751746035530LL;
  const long long sta_value = 34464302400860LL;
  const long long end_value = 2146196000086000LL;

  // const long long target = 35430LL;
  // const long long sta_value = 0LL;
  // const long long end_value = LONG_LONG_MAX;

  // CUDA configuration
  const int BLOCK_SIZE = 1024;
  const int NUM_BLOCKS = 8192;
  int VALUES_PER_THREAD = 10000;

  bool *d_found;
  long long *d_result;
  CUDA_CHECK(hipMalloc(&d_found, sizeof(bool)));
  CUDA_CHECK(hipMalloc(&d_result, sizeof(long long)));

  bool h_found = false;
  long long h_result = 0;

  // Initialize device memory
  CUDA_CHECK(hipMemset(d_found, 0, sizeof(bool)));

  printf("Starting GPU search from value: %lld to %lld\n", sta_value,
         end_value);

  long long current_start = sta_value;
  while (!h_found && current_start < end_value) {
    // Calculate remaining values to process
    long long batch_size =
        (long long)BLOCK_SIZE * NUM_BLOCKS * VALUES_PER_THREAD;
    long long remaining = end_value - current_start;
    if (remaining < batch_size) {
      // Adjust VALUES_PER_THREAD for the last batch
      VALUES_PER_THREAD = (int)((remaining + (BLOCK_SIZE * NUM_BLOCKS - 1)) /
                                (BLOCK_SIZE * NUM_BLOCKS));
    }

    searchKernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(
        current_start, target, VALUES_PER_THREAD, d_found, d_result);
    CUDA_CHECK(hipGetLastError());      // Check for launch errors
    CUDA_CHECK(hipDeviceSynchronize()); // Check for execution errors

    // Check if we found the result
    CUDA_CHECK(
        hipMemcpy(&h_found, d_found, sizeof(bool), hipMemcpyDeviceToHost));
    if (h_found) {
      CUDA_CHECK(hipMemcpy(&h_result, d_result, sizeof(long long),
                            hipMemcpyDeviceToHost));
      printf("Found result: %lld\n", h_result);
      break;
    }

    // Move to next batch
    current_start += (long long)BLOCK_SIZE * NUM_BLOCKS * VALUES_PER_THREAD;

    // Print progress every iteration
    printf("Processed up to: %lld (%.2f%%)\n", current_start,
           ((double)(current_start - sta_value) / (end_value - sta_value)) *
               100.0);
    fflush(stdout); // Force output to be displayed immediately
  }

  // Cleanup
  CUDA_CHECK(hipFree(d_found));
  CUDA_CHECK(hipFree(d_result));

  return 0;
}
